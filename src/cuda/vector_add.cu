#include "hip/hip_runtime.h"
#include "cuda/vector_add.h"

#include <cstdio>

#include "cuda/utils.h"
#include "mpi/utils.h"

#define THREADS_PER_BLOCK 512

__global__ void vectorAdd(float *Md, float *Nd, float *Pd, int width);

void cudaVectorAdd(float* chunk1, float* chunk2, float* chunkSum, int floatsPerNode) {
  float *gpuChunk1, *gpuChunk2, *gpuChunkSum;
  CUDA_CHECK(hipMalloc((void**) &gpuChunk1, floatsPerNode * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**) &gpuChunk2, floatsPerNode * sizeof(float)));
  CUDA_CHECK(hipMalloc((void**) &gpuChunkSum, floatsPerNode * sizeof(float)));

  CUDA_CHECK(hipMemcpy(gpuChunk1, chunk1, floatsPerNode * sizeof(float), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(gpuChunk2, chunk2, floatsPerNode * sizeof(float), hipMemcpyHostToDevice));

  int threadsPerBlock = THREADS_PER_BLOCK;
  int blocksPerGrid = floatsPerNode / THREADS_PER_BLOCK;
  if (floatsPerNode % THREADS_PER_BLOCK != 0)
    blocksPerGrid++;

  vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(gpuChunk1, gpuChunk2, gpuChunkSum, floatsPerNode);

  CUDA_CHECK(hipMemcpy(chunkSum, gpuChunkSum, floatsPerNode * sizeof(float), hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(gpuChunk1));
  CUDA_CHECK(hipFree(gpuChunk2));
  CUDA_CHECK(hipFree(gpuChunkSum));
}

__global__ void vectorAdd(float *Md, float *Nd, float *Pd, int width) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  while (tid < width) {
    Pd[tid] = Md[tid] + Nd[tid];
    tid += blockDim.x * gridDim.x;
  }
}
